#include "hip/hip_runtime.h"
__shared__ union {
  typename Load::TempStorage load;
  typename LoadFloat::TempStorage loadf;
  typename BlockReduce::TempStorage reduce;
} temp_storage;

for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
{
  valid_items = n - i >= (BLOCK_SIZE) ? (BLOCK_SIZE) : n - i;

  __syncthreads();
  Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items, 0.0f);
  __syncthreads();
  LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items, 0.0f);
  __syncthreads();
  LoadFloat(temp_storage.loadf).Load(&(state2[i]), s2_vals, valid_items, 0.0f);

  # pragma unroll NUM_VALS
  for(unsigned int j = 0; j < NUM_VALS; j++)
    g_vals[j] = gnorm_scale*((float)g_vals[j]);

  # pragma unroll NUM_VALS
  for(unsigned int j = 0; j < NUM_VALS; j++)
  {
    switch(OPTIMIZER)
    {
      case ADAM:
        s1_vals[j] = s1_vals[j]*beta1 + ((1.0f -beta1)*((float)g_vals[j]));
        s2_vals[j] = s2_vals[j]*beta2 + ((1.0f -beta2)*(((float)g_vals[j])*((float)g_vals[j])));
        s1_vals[j] *= correction1;
        s2_vals[j] *= correction2;
        s1_vals[j] = s1_vals[j]/(sqrtf(s2_vals[j])+eps); // update
        s1_vals[j] *= s1_vals[j]; // update l2 norm (update*update)
        break;
      case ADAMA:
        s1_vals[j] = s1_vals[j]*beta1 + ((1.0f -beta1)*((float)g_vals[j]));
        s2_vals[j] = s2_vals[j]*beta2 + ((1.0f -beta2)*(((float)g_vals[j])*((float)g_vals[j]))); // squared version
        s1_vals[j] *= correction1;
        s2_vals[j] *= correction2;
        s1_vals[j] = s1_vals[j]/(sqrtf(s2_vals[j])+eps); // update
        s1_vals[j] = sqrtf(s1_vals[j]); // update l2 norm (square root of update)
        break;
    }
  }

  __syncthreads();
  StoreT(storet).Store(&(p[i]), s1_vals, valid_items);
  __syncthreads();
  StoreT(storet).Store(&(unorm[i]), s2_vals, valid_items);
  __syncthreads();

  for(unsigned int j = 0; j < valid_items; j++)
  {
    p[i+j] -= lr*(s1_vals[j] + weight_decay*p[i+j]);
  }
}
__syncthreads();
Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items, 0.0f);
__syncthreads();
LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items, 0.0f);
__syncthreads();
LoadFloat(temp_storage.loadf).Load(&(state2[i]), s2_vals, valid_items, 0.0f);

# pragma unroll NUM_VALS
for(unsigned int j = 0; j < NUM_VALS; j++)
  g_vals[j] = gnorm_scale*((float)g_vals[j]);

# pragma unroll NUM_VALS
for(unsigned int j = 0; j < NUM_VALS; j++)
{
  switch(OPTIMIZER)
  {
    case ADAM:
      s1_vals[j] = s1_vals[j]*beta1 + ((1.0f -beta1)*((float)g_vals[j]));
      s2_vals[j] = s2_vals[j]*beta2 + ((1.0f -beta2)*(((float)g_vals[j])*((float)g_vals[j])));
      s1_vals[j] *= correction1;
      s2_vals[j] *= correction2;
      s1_vals[j] = s1_vals[j]/(sqrtf(s2_vals[j])+eps); // update
      s1_vals[j] *= s1_vals[j]; // update l2 norm (update*update)
      break;
    case ADAMA:
      s1_vals[j] = s1_vals[j]*beta1 + ((1.0f -beta1)*((float)g_vals[j]));
      s2_vals[j] = s2_vals[j]*beta2 + ((1.0f -beta2)*(((float)g_vals[j])*((float)g_vals[j]))); // squared version
      s1_vals[j] *= correction1;
      s2_vals[j] *= correction2;
      s1_vals[j] = s1_vals[j]/(sqrtf(s2_vals[j])+eps); // update
      s1_vals[j] = sqrtf(s1_vals[j]); // update l2 norm (square root of update)
      break;
  }
}

__syncthreads();
StoreT(storet).Store(&(p[i]), s1_vals, valid_items);
__syncthreads();
StoreT(storet).Store(&(unorm[i]), s2_vals, valid_items);
__syncthreads();

for(unsigned int j = 0; j < valid_items; j++)
{
  p[i+j] -= lr*(s1_vals[j] + weight_decay*p[i+j]);
}

# pragma unroll NUM_VALS-1
for(unsigned int j = 1; j < NUM_VALS; j++)
  s1_vals[0] += s1_vals[j];

__syncthreads();
s1_vals[0] = BlockReduce(temp_storage.reduce).Sum(s1_vals[0]);

if(threadIdx.x == 0)
  atomicAdd(&unorm[0], s1_vals[0]);

__syncwarp();
}